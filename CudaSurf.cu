#include "hip/hip_runtime.h"
/*MIT License

Copyright (c) 2019 Xavier Martinez

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include "hip/hip_runtime.h"

// includes
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <sstream>
#include <iterator>
#include <memory>
#include <map>

// #include <cassert>
#include <fstream>
#include <algorithm>
#include <functional>
#include <cctype>
#include <locale>
#include <math.h>

#include <vector>

#include "args.hxx"

#include "Kernels.cu"
#include "cpdb.h"
#include "SmoothMesh.h"
#include "CudaSurf.h"

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>
#include <thrust/binary_search.h>
#include <hip/hip_runtime.h>
using namespace std;

int SLICE = 300;
float probeRadius = PROBERADIUS;
float gridResolutionNeighbor;
float gridResolutionSES = 0.5f;
int laplacianSmoothSteps = 1;
string outputFilePath = "output.obj";
string inputFilePath = "";

unsigned int getMinMax(chain *C, float3 *minVal, float3 *maxVal, float *maxAtom)
{
    atom *A = NULL;
    unsigned int N = 0;

    A = &C->residues[0].atoms[0];
    float3 vmin, vmax, coords;

    vmin.x = vmin.y = vmin.z = 100000.0f;
    vmax.x = vmax.y = vmax.z = -100000.0f;
    *maxAtom = 0.0f;
    while (A != NULL)
    {
        coords = A->coor;
        vmin.x = std::min(vmin.x, coords.x);
        vmin.y = std::min(vmin.y, coords.y);
        vmin.z = std::min(vmin.z, coords.z);

        vmax.x = std::max(vmax.x, coords.x);
        vmax.y = std::max(vmax.y, coords.y);
        vmax.z = std::max(vmax.z, coords.z);

        float atomRad;
        if (radiusDic.count(A->element[0]))
            atomRad = radiusDic[A->element[0]];
        else
            atomRad = radiusDic['X'];
        *maxAtom = std::max(*maxAtom, atomRad);
        N++;
        A = A->next;
    }
    *minVal = vmin;
    *maxVal = vmax;
    return N;
}
unsigned int getMinMax(pdb *P, float3 *minVal, float3 *maxVal, float *maxAtom)
{
    atom *A = NULL;
    unsigned int N = 0;
    chain *C = NULL;
    *maxAtom = 0.0f;
    float3 vmin, vmax, coords;

    vmin.x = vmin.y = vmin.z = 100000.0f;
    vmax.x = vmax.y = vmax.z = -100000.0f;

    for (int chainId = 0; chainId < P->size; chainId++)
    {
        C = &P->chains[chainId];

        A = &C->residues[0].atoms[0];

        while (A != NULL)
        {
            coords = A->coor;
            vmin.x = std::min(vmin.x, coords.x);
            vmin.y = std::min(vmin.y, coords.y);
            vmin.z = std::min(vmin.z, coords.z);

            vmax.x = std::max(vmax.x, coords.x);
            vmax.y = std::max(vmax.y, coords.y);
            vmax.z = std::max(vmax.z, coords.z);

            float atomRad;
            if (radiusDic.count(A->element[0]))
                atomRad = radiusDic[A->element[0]];
            else
                atomRad = radiusDic['X'];
            *maxAtom = std::max(*maxAtom, atomRad);
            N++;
            A = A->next;
        }
    }
    *minVal = vmin;
    *maxVal = vmax;
    return N;
}
// center of atom
void getMinMax(float3 *positions, float *radii, unsigned int N, float3 *minVal, float3 *maxVal, float *maxAtom)
{
    *maxAtom = 0.0f;
    float3 vmin, vmax, coords;

    vmin.x = vmin.y = vmin.z = 100000.0f;
    vmax.x = vmax.y = vmax.z = -100000.0f;

    for (unsigned int a = 0; a < N; a++)
    {
        coords = positions[a];
        vmin.x = std::min(vmin.x, coords.x);
        vmin.y = std::min(vmin.y, coords.y);
        vmin.z = std::min(vmin.z, coords.z);

        vmax.x = std::max(vmax.x, coords.x);
        vmax.y = std::max(vmax.y, coords.y);
        vmax.z = std::max(vmax.z, coords.z);

        float atomRad = radii[a];

        *maxAtom = std::max(*maxAtom, atomRad);
    }
    *minVal = vmin;
    *maxVal = vmax;
}

float4 *getArrayAtomPosRad(chain *C, unsigned int N)
{

    float4 *result = new float4[N];
    atom *A = NULL;
    int id = 0;

    A = &C->residues[0].atoms[0];
    float3 coords;
    while (A != NULL)
    {
        coords = A->coor;

        float atomRad = radiusDic[A->element[0]];
        result[id].x = coords.x;
        result[id].y = coords.y;
        result[id].z = coords.z;
        result[id].w = atomRad;
        id++;
        A = A->next;
    }

    return result;
}

float4 *getArrayAtomPosRad(pdb *P, unsigned int N)
{
    chain *C = NULL;
    atom *A = NULL;
    float4 *result = new float4[N];
    // float4 *result;
    int id = 0;

    for (int chainId = 0; chainId < P->size; chainId++)
    {
        C = &P->chains[chainId];

        A = &C->residues[0].atoms[0];
        float3 coords;
        while (A != NULL)
        {
            coords = A->coor;

            float atomRad = radiusDic[A->element[0]];
            result[id].x = coords.x;
            result[id].y = coords.y;
            result[id].z = coords.z;
            result[id].w = atomRad;
            id++;
            A = A->next;
        }
    }

    return result;
}

float4 *getArrayAtomPosRad(float3 *positions, float *radii, unsigned int N)
{
    float4 *result = (float4 *)malloc(sizeof(float4) * N);
    int id = 0;

    for (int a = 0; a < N; a++)
    {
        float3 coords = positions[a];
        float atomRad = radii[a];
        result[id].x = coords.x;
        result[id].y = coords.y;
        result[id].z = coords.z;
        result[id].w = atomRad;
        id++;
    }

    return result;
}

float computeMaxDist(float3 minVal, float3 maxVal, float maxAtomRad)
{
    return std::max(maxVal.x - minVal.x, std::max(maxVal.y - minVal.y, maxVal.z - minVal.z)) + (2 * maxAtomRad) + (4 * probeRadius);
}

void writeToObj(const string &fileName, const vector<int> &meshTriSizes, const vector<int> &meshVertSizes,
                const vector<float3 *> &Allvertices, const vector<int3 *> &AllTriangles)
{

#if MEASURETIME
    std::clock_t start = std::clock();
#endif

    FILE *fptr;
    if ((fptr = fopen(fileName.c_str(), "w")) == NULL)
    {
        fprintf(stderr, "Failed to open output file\n");
        exit(-1);
    }
    for (int m = 0; m < meshTriSizes.size(); m++)
    {

        for (int i = 0; i < meshVertSizes[m]; i++)
        {
            float3 vert = Allvertices[m][i];
            fprintf(fptr, "v %.3f %.3f %.3f\n", vert.x, vert.y, vert.z);
        }
    }

    fprintf(fptr, "\n");
    unsigned int cumulMesh = 0;
    for (int m = 0; m < meshTriSizes.size(); m++)
    {
        int ntri = meshTriSizes[m];
        for (int i = 0; i < ntri; i++)
        {
            if (AllTriangles[m][i].x != AllTriangles[m][i].y && AllTriangles[m][i].x != AllTriangles[m][i].z && AllTriangles[m][i].y != AllTriangles[m][i].z)
            {
                fprintf(fptr, "f %d %d %d\n", cumulMesh + AllTriangles[m][i].y + 1, cumulMesh + AllTriangles[m][i].x + 1, cumulMesh + AllTriangles[m][i].z + 1);
            }
        }
        cumulMesh += meshVertSizes[m];
    }

    fclose(fptr);

#if MEASURETIME
    std::cerr << "Time for writting " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
#endif
}

void writeToObj(const string &fileName, const MeshData &mesh)
{
#if MEASURETIME
    std::clock_t start = std::clock();
#endif

    FILE *fptr;
    if ((fptr = fopen(fileName.c_str(), "w")) == NULL)
    {
        fprintf(stderr, "Failed to open output file\n");
        exit(-1);
    }

    for (int i = 0; i < mesh.NVertices; i++)
    {
        float3 vert = mesh.vertices[i];
        fprintf(fptr, "v %.3f %.3f %.3f\n", vert.x, vert.y, vert.z);
    }

    fprintf(fptr, "\n");
    for (int i = 0; i < mesh.NTriangles; i++)
    {
        fprintf(fptr, "f %d %d %d\n", mesh.triangles[i].y + 1, mesh.triangles[i].x + 1, mesh.triangles[i].z + 1);
    }
    fclose(fptr);
#if MEASURETIME
    std::cerr << "Time for writting " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
#endif
}
void writeToObj(const string &fileName, std::vector<MeshData> meshes)
{

#if MEASURETIME
    std::clock_t start = std::clock();
#endif

    FILE *fptr;
    if ((fptr = fopen(fileName.c_str(), "w")) == NULL)
    {
        fprintf(stderr, "Failed to open output file\n");
        exit(-1);
    }
    unsigned int cumulVert = 0;
    for (int m = 0; m < meshes.size(); m++)
    {
        MeshData mesh = meshes[m];

        // smoothMeshLaplacian(2, mesh);

        for (int i = 0; i < mesh.NVertices; i++)
        {
            float3 vert = mesh.vertices[i];
            fprintf(fptr, "v %.3f %.3f %.3f\n", vert.x, vert.y, vert.z);
        }
    }
    fprintf(fptr, "\n");
    for (int m = 0; m < meshes.size(); m++)
    {
        MeshData mesh = meshes[m];

        for (int i = 0; i < mesh.NTriangles; i++)
        {
            fprintf(fptr, "f %d %d %d\n", cumulVert + mesh.triangles[i].y + 1, cumulVert + mesh.triangles[i].x + 1, cumulVert + mesh.triangles[i].z + 1);
        }
        cumulVert += mesh.NVertices;
    }
    fclose(fptr);
#if MEASURETIME
    std::cerr << "Time for writting " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
#endif
}

MeshData computeMarchingCubes(int3 sliceGridSESDim, int cutMC, int sliceNbCellSES, float *cudaGridValues, uint2 *vertPerCell,
                              unsigned int *compactedVoxels, int3 gridSESDim, float4 originGridSESDx, int3 offset, float4 *cudaSortedAtomPosRad,
                              int2 *cellStartEnd, int3 gridNeighborDim, float4 originGridNeighborDx, int rangeSearchRefine)
{

    unsigned long int memAlloc = 0;
    memsetCudaUInt2<<<(sliceNbCellSES + NBTHREADS - 1) / NBTHREADS, NBTHREADS>>>(vertPerCell, make_uint2(0, 0), sliceNbCellSES);

    MeshData result;
    float iso = 0.0f;
    dim3 localWorkSize(cutMC, cutMC, cutMC);
    dim3 globalWorkSize((sliceGridSESDim.x + cutMC - 1) / cutMC, (sliceGridSESDim.y + cutMC - 1) / cutMC, (sliceGridSESDim.z + cutMC - 1) / cutMC);

    // 每个ses grid 形成了多少个三角形面片
    countVertexPerCell<<<globalWorkSize, localWorkSize>>>(iso, sliceGridSESDim, cudaGridValues, vertPerCell, rangeSearchRefine, offset);
    gpuErrchk(hipPeekAtLastError());

    uint2 lastElement, lastScanElement;
    gpuErrchk(hipMemcpy((void *)&lastElement, (void *)(vertPerCell + sliceNbCellSES - 1), sizeof(uint2), hipMemcpyDeviceToHost));

    // https:// thrust.github.io/doc/group__prefixsums_ga7be5451c96d8f649c8c43208fcebb8c3.html 看例子
    thrust::exclusive_scan(thrust::device_ptr<uint2>(vertPerCell),
                           thrust::device_ptr<uint2>(vertPerCell + sliceNbCellSES),
                           thrust::device_ptr<uint2>(vertPerCell),
                           make_uint2(0, 0), add_uint2());

    gpuErrchk(hipMemcpy((void *)&lastScanElement, (void *)(vertPerCell + sliceNbCellSES - 1), sizeof(uint2), hipMemcpyDeviceToHost));

    // exclusive_scan最后一个数不会加进去，所以得手动
    unsigned int totalVoxels = lastElement.y + lastScanElement.y;
    unsigned int totalVerts = lastElement.x + lastScanElement.x;

    float3 *cudaVertices;
    gpuErrchk(hipMalloc(&cudaVertices, sizeof(float3) * totalVerts));
    memAlloc += sizeof(float3) * totalVerts;

    globalWorkSize = dim3((sliceGridSESDim.x + localWorkSize.x - 1) / localWorkSize.x, (sliceGridSESDim.y + localWorkSize.y - 1) / localWorkSize.y, (sliceGridSESDim.z + localWorkSize.z - 1) / localWorkSize.z);

    compactVoxels<<<globalWorkSize, localWorkSize>>>(compactedVoxels, vertPerCell, lastElement.y, sliceNbCellSES, sliceNbCellSES + 1, sliceGridSESDim, rangeSearchRefine, offset);
    gpuErrchk(hipPeekAtLastError());

    unsigned int totalVoxsqr3 = (unsigned int)ceil((totalVoxels + NBTHREADS - 1) / NBTHREADS);
    globalWorkSize = dim3(totalVoxsqr3, 1, 1);
    if (totalVoxsqr3 == 0)
    {
        return result;
    }

    generateTriangleVerticesSMEM<<<globalWorkSize, NBTHREADS>>>(cudaVertices, compactedVoxels, vertPerCell, cudaGridValues, originGridSESDx,
                                                                iso, totalVoxels, totalVerts - 3, sliceGridSESDim, offset);

    gpuErrchk(hipPeekAtLastError());

    // Weld vertices
    float3 *vertOri;
    int *cudaTri;
    int *cudaAtomIdPerVert;

    int global = (unsigned int)ceil((totalVerts + NBTHREADS - 1) / NBTHREADS);
    groupVertices<<<global, NBTHREADS>>>(cudaVertices, totalVerts, EPSILON);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMalloc(&vertOri, sizeof(float3) * totalVerts));
    gpuErrchk(hipMemcpy(vertOri, cudaVertices, sizeof(float3) * totalVerts, hipMemcpyDeviceToDevice));
    gpuErrchk(hipMalloc(&cudaTri, sizeof(int) * totalVerts));

    memAlloc += sizeof(float3) * totalVerts;
    memAlloc += sizeof(int) * totalVerts;

    thrust::device_ptr<float3> vertThrust(cudaVertices);
    thrust::sort(vertThrust, vertThrust + totalVerts, sort_float3());

    thrust::device_ptr<float3> last = thrust::unique(vertThrust, vertThrust + totalVerts, samefloat3());

    unsigned int newtotalVerts = last - vertThrust;

    thrust::device_ptr<float3> vertOriThrust(vertOri);
    thrust::device_ptr<int> triThrust(cudaTri);
    thrust::lower_bound(vertThrust, last, vertOriThrust, vertOriThrust + totalVerts, triThrust, lessf3<float3>());
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMalloc(&cudaAtomIdPerVert, sizeof(int) * newtotalVerts));
    memAlloc += sizeof(int) * newtotalVerts;

    global = (unsigned int)ceil((newtotalVerts + NBTHREADS - 1) / NBTHREADS);

    // Look for atoms around vertices => could be done a way smarter way during the MC step
    closestAtomPerVertex<<<global, NBTHREADS>>>(cudaAtomIdPerVert, cudaVertices, newtotalVerts, gridNeighborDim,
                                                originGridNeighborDx, originGridSESDx, cellStartEnd, cudaSortedAtomPosRad);

    gpuErrchk(hipPeekAtLastError());

    cerr << "MC allocation = " << memAlloc / 1000000.0f << " Mo" << endl;

    int Ntriangles = totalVerts / 3;

    result.vertices = (float3 *)malloc(sizeof(float3) * newtotalVerts);
    result.triangles = (int3 *)malloc(sizeof(int3) * Ntriangles);
    result.atomIdPerVert = (int *)malloc(sizeof(int) * newtotalVerts);
    result.NVertices = newtotalVerts;
    result.NTriangles = Ntriangles;

    int *tmpTri = (int *)malloc(sizeof(int) * totalVerts);

    gpuErrchk(hipMemcpy(result.vertices, cudaVertices, sizeof(float3) * newtotalVerts, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(result.atomIdPerVert, cudaAtomIdPerVert, sizeof(int) * newtotalVerts, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(tmpTri, cudaTri, sizeof(int) * totalVerts, hipMemcpyDeviceToHost));

    // Store the triangle in a 3d vector
    for (int i = 0; i < Ntriangles; i++)
    {
        result.triangles[i].x = tmpTri[i * 3 + 0];
        result.triangles[i].y = tmpTri[i * 3 + 1];
        result.triangles[i].z = tmpTri[i * 3 + 2];
    }
    free(tmpTri);

    gpuErrchk(hipFree(cudaVertices));
    gpuErrchk(hipFree(vertOri));
    gpuErrchk(hipFree(cudaTri));
    gpuErrchk(hipFree(cudaAtomIdPerVert));

    return result;
}

std::vector<MeshData> computeSlicedSES(float3 positions[], float radii[], unsigned int N, float resoSES, int doSmoothing = 1)
{
#if MEASURETIME
    std::clock_t startSES = std::clock();
#endif

    // Record a mesh per slice
    std::vector<MeshData> resultMeshes;

    float3 minVal, maxVal;
    float maxAtomRad = 0.0;

    getMinMax(positions, radii, N, &minVal, &maxVal, &maxAtomRad);

    cerr << "#atoms : " << N << endl;
    if (N <= 1)
    {
        cerr << "Failed to parse the PDB or empty PDB file" << endl;
        return resultMeshes;
    }

    float4 *atomPosRad = getArrayAtomPosRad(positions, radii, N);
    float maxDist = computeMaxDist(minVal, maxVal, maxAtomRad); // 格子的最长距离，max（bondingbox）+ max_atom_radius + probe的直径

    gridResolutionNeighbor = probeRadius + maxAtomRad;

    // Grid is a cube
    float3 originGridNeighbor = {
        minVal.x - maxAtomRad - 2 * probeRadius,
        minVal.y - maxAtomRad - 2 * probeRadius,
        minVal.z - maxAtomRad - 2 * probeRadius};

    int gridNeighborSize = (int)ceil(maxDist / gridResolutionNeighbor);

    int3 gridNeighborDim = {gridNeighborSize, gridNeighborSize, gridNeighborSize};

    int gridSESSize = (int)ceil(maxDist / resoSES);

    int3 gridSESDim = {gridSESSize, gridSESSize, gridSESSize};

    float4 originGridNeighborDx = {
        originGridNeighbor.x,
        originGridNeighbor.y,
        originGridNeighbor.z,
        gridResolutionNeighbor};

    float4 originGridSESDx = {
        originGridNeighborDx.x,
        originGridNeighborDx.y,
        originGridNeighborDx.z,
        resoSES};

    unsigned int nbcellsNeighbor = gridNeighborDim.x * gridNeighborDim.y * gridNeighborDim.z;
    // unsigned int nbcellsSES = gridSESDim.x * gridSESDim.y * gridSESDim.z;

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    float4 *cudaAtomPosRad;
    float4 *cudaSortedAtomPosRad;
    int2 *cudaHashIndex;
    int2 *cellStartEnd;
    float *cudaGridValues; // sdf 定义的 值
    int *cudaFillCheck;

    // Marching cubes data
    uint2 *vertPerCell;
    unsigned int *compactedVoxels;

    gpuErrchk(hipMalloc((void **)&cudaAtomPosRad, sizeof(float4) * N));
    gpuErrchk(hipMalloc((void **)&cudaSortedAtomPosRad, sizeof(float4) * N));
    gpuErrchk(hipMalloc((void **)&cudaHashIndex, sizeof(int2) * N));
    gpuErrchk(hipMalloc((void **)&cellStartEnd, sizeof(int2) * nbcellsNeighbor));

    //-------------- Step 1 : Insert atoms in neighbor cells -----------------

    // Copy atom positions and radii to GPU
    gpuErrchk(hipMemcpy(cudaAtomPosRad, atomPosRad, sizeof(float4) * N, hipMemcpyHostToDevice));

    // Compute atom cell ids
    hashAtoms<<<N, NBTHREADS>>>(N, cudaAtomPosRad, gridNeighborDim, originGridNeighborDx, cudaHashIndex, N);

    gpuErrchk(hipPeekAtLastError());

    // Sort atoms cell id
    compare_int2 cmp;
    thrust::device_ptr<int2> D_beg = thrust::device_pointer_cast(cudaHashIndex);
    thrust::sort(D_beg, D_beg + N, cmp);
    gpuErrchk(hipPeekAtLastError());

    memsetCudaInt2<<<(nbcellsNeighbor + NBTHREADS - 1) / NBTHREADS, NBTHREADS>>>(cellStartEnd, make_int2(EMPTYCELL, EMPTYCELL), nbcellsNeighbor);

    // Reorder atoms positions and radii and fill cellStartEnd
    // 按照所处的cell index 把 atoms 排序， 然后记录 每个 cell 所占atom的编号，从哪开始 到哪儿了介绍
    sortCell<<<N, NBTHREADS>>>(N, cudaAtomPosRad, cudaHashIndex, cudaSortedAtomPosRad, cellStartEnd);

    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipFree(cudaAtomPosRad));

    // std::cerr << "Time for setup " << (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
    // start = std::clock();

    //-------------- Step 2 : Compute points of the grid outside or inside the surface -----------------
    // Use slices of the grid to avoid allocating large amount of data
    // 具体实现的时候，不考虑voxel的体积，把它当作一个点(i,j,k)来求距离
    // offset是为了保证所有的sub-grid都会遍历到，包括余出来的
    int rangeSearchRefine = (int)ceil(PROBERADIUS / resoSES);
    int sliceSmallSize = min(SLICE, gridSESSize);
    int sliceSize = min(SLICE + 2 * rangeSearchRefine, gridSESSize);
    // int sliceSmallNbCellSES = sliceSmallSize * sliceSmallSize * sliceSmallSize;
    int sliceNbCellSES = sliceSize * sliceSize * sliceSize;
    // int3 sliceGridSESDim = make_int3(sliceSmallSize, sliceSmallSize, sliceSmallSize);
    int3 fullSliceGridSESDim = make_int3(sliceSize, sliceSize, sliceSize);

    gpuErrchk(hipMalloc((void **)&cudaGridValues, sizeof(float) * sliceNbCellSES));
    gpuErrchk(hipMalloc((void **)&cudaFillCheck, sizeof(int) * sliceNbCellSES));

    gpuErrchk(hipMalloc(&vertPerCell, sizeof(uint2) * sliceNbCellSES));
    gpuErrchk(hipMalloc(&compactedVoxels, sizeof(unsigned int) * sliceNbCellSES));

    gpuErrchk(hipPeekAtLastError());

    cerr << "Allocating " << (((sizeof(int) + sizeof(float)) * sliceNbCellSES + 3 * sizeof(int) * sliceNbCellSES) + 2 * sizeof(float4) * N + sizeof(int2) * N + sizeof(int2) * nbcellsNeighbor) / 1000000.0f << " Mo" << endl;

    int3 offset = {0, 0, 0};
    int cut = 8;

    cerr << "Full size grid = " << gridSESSize << " x " << gridSESSize << " x " << gridSESSize << endl;
    // hipEventRecord(start);
    // for (int slice = 0; slice < gridSESSize; slice += sliceSmallSize) {
    for (int i = 0; i < gridSESSize; i += sliceSmallSize)
    {
        offset.x = i;
        for (int j = 0; j < gridSESSize; j += sliceSmallSize)
        {
            offset.y = j;
            for (int k = 0; k < gridSESSize; k += sliceSmallSize)
            {
                offset.z = k;
                // cerr << "-----------------------------\nStarting : " << offset.x << " / " << offset.y << " / " << offset.z << endl;

                memsetCudaFloat<<<(sliceNbCellSES + NBTHREADS - 1) / NBTHREADS, NBTHREADS>>>(cudaGridValues, probeRadius, sliceNbCellSES);
                memsetCudaInt<<<(sliceNbCellSES + NBTHREADS - 1) / NBTHREADS, NBTHREADS>>>(cudaFillCheck, EMPTYCELL, sliceNbCellSES);

                dim3 localWorkSize(cut, cut, cut);
                // dim3 globalWorkSize((sliceSmallSize + cut - 1) / cut, (sliceSmallSize + cut - 1) / cut, (sliceSmallSize + cut - 1) / cut);
                dim3 globalWorkSize((sliceSize + cut - 1) / cut, (sliceSize + cut - 1) / cut, (sliceSize + cut - 1) / cut);

                int3 reducedOffset = make_int3(max(0, offset.x - rangeSearchRefine),
                                               max(0, offset.y - rangeSearchRefine),
                                               max(0, offset.z - rangeSearchRefine));

                // int3 reducedOffset = offset;

                // cerr << "Fulllll : " << fullSliceGridSESDim.x << ", " << fullSliceGridSESDim.y << ", " << fullSliceGridSESDim.z << endl;
                // cerr << "global = " << globalWorkSize.x << ", " << globalWorkSize.y << ", " << globalWorkSize.z << "   " << (sliceSmallSize + cut - 1) / cut << endl;

                probeIntersection<<<globalWorkSize, localWorkSize>>>(cudaFillCheck, cudaHashIndex, gridNeighborDim, originGridNeighborDx,
                                                                     gridSESDim, fullSliceGridSESDim, originGridSESDx, cellStartEnd,
                                                                     cudaSortedAtomPosRad, cudaGridValues, /*offset*/ reducedOffset, N, sliceNbCellSES);

                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());

                // Count cells at the border, cells that will be used in the refinement step
                thrust::device_ptr<int> fillThrust(cudaFillCheck);
                thrust::sort(fillThrust, fillThrust + sliceNbCellSES);

                unsigned int notEmptyCells = thrust::count_if(thrust::device, fillThrust, fillThrust + sliceNbCellSES, is_notempty());

                if (notEmptyCells == 0)
                {
                    // cerr << "Empty cells !!!" << endl;
                    continue;
                }

                localWorkSize = dim3(NBTHREADS, 1.0f, 1.0f);

                // Too long execution of this kernel triggers the watchdog timer => cut it
                int tranche = min(notEmptyCells, 65536 / 8 * NBTHREADS);

                const int nbStream = 4;
                hipStream_t streams[nbStream];
                for (int i = 0; i < nbStream; i++)
                    hipStreamCreate(&(streams[i]));
                int idStream = 0;

                for (unsigned int o = 0; o < notEmptyCells; o += tranche)
                {

                    globalWorkSize = dim3((tranche + NBTHREADS - 1) / NBTHREADS, 1.0f, 1.0f);
                    // cerr <<o<< " Launch (" << globalWorkSize.x << ", "<<globalWorkSize.y<<", "<<globalWorkSize.z<<") x ("<<localWorkSize.x<<", "<<localWorkSize.y<<", 1.0)" << endl;

                    distanceFieldRefine<<<globalWorkSize, localWorkSize, 0, streams[idStream]>>>(cudaFillCheck, cudaHashIndex, gridNeighborDim, originGridNeighborDx,
                                                                                                 gridSESDim, fullSliceGridSESDim, originGridSESDx, cellStartEnd,
                                                                                                 cudaSortedAtomPosRad, cudaGridValues, N, notEmptyCells, reducedOffset, o);

                    idStream++;
                    if (idStream == nbStream)
                        idStream = 0;
                }

                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());

                for (int i = 0; i < nbStream; i++)
                    hipStreamDestroy(streams[i]);

                // Reset grid values that are outside of the slice

                // localWorkSize = dim3(cut, cut, cut);
                // globalWorkSize = dim3((sliceSize + cut - 1) / cut, (sliceSize + cut - 1) / cut, (sliceSize + cut - 1) / cut);

                // resetGridValuesSlice <<< globalWorkSize, localWorkSize >>> (offset, rangeSearchRefine - 1, fullSliceGridSESDim, cudaGridValues);

                // Marching cubes
                MeshData mesh = computeMarchingCubes(fullSliceGridSESDim, cut, sliceNbCellSES, cudaGridValues,
                                                     vertPerCell, compactedVoxels, gridSESDim, originGridSESDx, reducedOffset,
                                                     cudaSortedAtomPosRad, cellStartEnd, gridNeighborDim, originGridNeighborDx, rangeSearchRefine);

                smoothMeshLaplacian(doSmoothing, mesh);
                resultMeshes.push_back(mesh);

                // if(resultMeshes.size() == 2){
                // return resultMeshes;
                // }
                // break;
            }
            // break;
        }
        // break;
    }
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cerr << "Time for step 2 : " << milliseconds << " ms" << std::endl;

    hipFree(cudaSortedAtomPosRad);
    hipFree(cudaHashIndex);
    hipFree(cellStartEnd);
    hipFree(cudaGridValues);
    hipFree(cudaFillCheck);
    hipFree(vertPerCell);
    hipFree(compactedVoxels);

    free(atomPosRad);

#if MEASURETIME
    std::cerr << "Time for computing SES " << (std::clock() - startSES) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;
#endif

    return resultMeshes;
}

extern "C"
{
    int NTriangles;
    int NVertices;
    int *globalTriangles;
    float3 *globalVertices;
    int *globalIdAtomPerVert;
}

API void API_computeSES(float resoSES, float3 *atomPos, float *atomRad, unsigned int N, float3 *out_vertices,
                        unsigned int *NVert, int *out_triangles, unsigned int *NTri, int doSmoothing)
{

    // float3 *positions = (float3 *)malloc(sizeof(float3) * N);

    // for (int a = 0; a < N; a++) {
    //     positions[a] = make_float3(atomPos[a * 3 + 0], atomPos[a * 3 + 1], atomPos[a * 3 + 2]);
    // }

    *NVert = 0;
    *NTri = 0;

    std::vector<MeshData> resultMeshes = computeSlicedSES(atomPos, atomRad, N, resoSES, doSmoothing);

    unsigned int totalVerts = 0;
    unsigned int totalTris = 0;

    for (int i = 0; i < resultMeshes.size(); i++)
    {
        totalVerts += resultMeshes[i].NVertices;
        totalTris += resultMeshes[i].NTriangles * 3;
    }
    globalVertices = (float3 *)malloc(sizeof(float3) * totalVerts);
    globalTriangles = (int *)malloc(sizeof(int) * totalTris);
    globalIdAtomPerVert = (int *)malloc(sizeof(int) * totalVerts);

    unsigned int cumulVert = 0;
    unsigned int curIdV = 0;
    unsigned int curIdT = 0;

    for (int i = 0; i < resultMeshes.size(); i++)
    {
        for (int v = 0; v < resultMeshes[i].NVertices; v++)
        {
            globalVertices[curIdV] = resultMeshes[i].vertices[v];
            globalIdAtomPerVert[curIdV] = resultMeshes[i].atomIdPerVert[v];
            curIdV++;
        }
        for (int t = 0; t < resultMeshes[i].NTriangles; t++)
        {
            globalTriangles[curIdT++] = resultMeshes[i].triangles[t].x + cumulVert;
            globalTriangles[curIdT++] = resultMeshes[i].triangles[t].y + cumulVert;
            globalTriangles[curIdT++] = resultMeshes[i].triangles[t].z + cumulVert;
        }
        cumulVert += resultMeshes[i].NVertices;
    }

    *NVert = totalVerts;
    *NTri = totalTris;
    NTriangles = totalTris;
    NVertices = totalVerts;
    // free(positions);

    // globalVertices = out_vertices;
    // globalTriangles = out_triangles;
}

extern "C"
{
    API int *API_getTriangles(bool invertTriangles = false)
    {
        if (invertTriangles)
        {
            for (unsigned int t = 0; t < NTriangles / 3; t++)
            {
                int save = globalTriangles[t * 3];
                globalTriangles[t * 3] = globalTriangles[t * 3 + 1];
                globalTriangles[t * 3 + 1] = save;
            }
        }
        return globalTriangles;
    }
    API float3 *API_getVertices()
    {
        return globalVertices;
    }
    API int *API_getAtomIdPerVert()
    {
        return globalIdAtomPerVert;
    }

    API void API_freeMesh()
    {
        free(globalVertices);
        free(globalTriangles);
        free(globalIdAtomPerVert);
    }
}

int main(int argc, const char *argv[])
{

    args::ArgumentParser parser("QuickSES, SES mesh generation using GPU", "");
    args::Group groupMandatory(parser, "", args::Group::Validators::All);
    args::Group groupOptional(parser, "", args::Group::Validators::DontCare);
    args::ValueFlag<string> inFile(groupMandatory, "input.pdb", "Input PDB file", {'i'});
    args::ValueFlag<string> outFile(groupMandatory, "output.obj", "Output OBJ mesh file", {'o'});
    args::ValueFlag<int> smoothTimes(groupOptional, "smooth factor", "(1) Times to run Laplacian smoothing step.", {'l'});
    args::ValueFlag<float> voxelSize(groupOptional, "voxel size", "(0.5) Voxel size in Angstrom. Defines the quality of the mesh.", {'v'});
    args::ValueFlag<int> slice(groupOptional, "slice size", "(300) Size of the sub-grid. Defines the quantity of GPU memory needed.", {'s'});
    args::HelpFlag help(groupOptional, "help", "   Display this help menu", {'h', "help"});

    try
    {
        parser.ParseCLI(argc, argv);
    }
    catch (args::Help)
    {
        std::cerr << parser;
        return 0;
    }
    catch (args::ParseError e)
    {
        std::cerr << e.what() << std::endl;
        std::cerr << parser;
        return -1;
    }
    catch (args::ValidationError e)
    {
        // std::cerr << e.what() << std::endl;
        std::cerr << "Usage: " << parser;
        return -1;
    }

    if (inFile)
    {
        inputFilePath = args::get(inFile);
    }
    if (outFile)
    {
        outputFilePath = args::get(outFile);
    }
    if (smoothTimes)
    {
        laplacianSmoothSteps = args::get(smoothTimes);
    }
    if (voxelSize)
    {
        gridResolutionSES = args::get(voxelSize);
    }
    if (slice)
    {
        SLICE = args::get(slice);
    }

    std::clock_t startparse = std::clock();

    initRadiusDic();

    pdb *P;
    P = initPDB();

    parsePDB((char *)inputFilePath.c_str(), P, (char *)"");

    cerr << "Grid resolution = " << gridResolutionSES << endl;
    std::cerr << "Time for parse " << (std::clock() - startparse) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << std::endl;

    unsigned int N = 0;
    std::vector<float3> atomPos;
    std::vector<float> atomRadii;

    atom *A = NULL;
    chain *C = NULL;

    for (int chainId = 0; chainId < P->size; chainId++)
    {
        C = &P->chains[chainId];

        A = &C->residues[0].atoms[0];

        while (A != NULL)
        {
            float3 coords = A->coor;
            atomPos.push_back(coords);
            float atomRad;
            if (radiusDic.count(A->element[0]))
                atomRad = radiusDic[A->element[0]];
            else
                atomRad = radiusDic['X'];
            atomRadii.push_back(atomRad);

            N++;
            A = A->next;
        }
    }

    std::vector<MeshData> resultMeshes = computeSlicedSES(&atomPos[0], &atomRadii[0], N, gridResolutionSES, laplacianSmoothSteps);
    // std::vector<MeshData> resultMeshes = computeSlicedSESCPU(P);

    // Write to OBJ
    writeToObj(outputFilePath, resultMeshes);

    freePDB(P);

    return 0;
}
